#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
  hipDeviceProp_t prop;
  int count, devNum;
  hipGetDeviceCount(&count);
  printf(" Number of devices = %d\n", count);
  hipGetDevice(&devNum);
  printf(" Device Number = %d\n", devNum);

  for(int i = 0;i<count;i++) {
    hipGetDeviceProperties(&prop, i);
    printf("\n Name:%s", prop.name);
    printf("\n Clock Rate = %d", prop.clockRate);
    printf("\n Device copy overlap:");
    if(prop.deviceOverlap) { printf("\t Enabled");}
    else{printf("\t Disabled");}
    printf("\n Kernel execution Timeout:");
    if(prop.kernelExecTimeoutEnabled) {printf("Enabled\n");}
    else{printf("Disabled\n");}
    printf("\n Global Memory: %ld", prop.totalGlobalMem);
    printf("\n Constant Memory: %ld", prop.totalConstMem);
    printf("\n Max Texture MemID: %d", prop.maxTexture1D);
  }


  return;
}